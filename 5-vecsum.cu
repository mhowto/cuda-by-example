#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/index.h"

#define N	(33 * 1024)

__global__ void add( int *a, int *b, int *c ) {
	int tid = getGlobalIdx_1D_1D();	// handle the data at this index
	while (tid < N) {
	    c[tid] = a[tid] + b[tid];
	    tid += blockDim.x * gridDim.x;
	}
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the cpu
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	for( int i = 0; i < N; i++ ) {
		a[i] = -i;
		b[i] = i * i;
	}

	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

	add<<<128, 128>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

    bool success = true;
	for( int i = 0; i < N; i++ ){
	    if ((a[i] + b[i]) != c[i]) {
		    printf( "Error: %d + %d != %d\n", a[i], b[i], c[i] );
		    success =false;
	    }
	}

	if (success) printf("We did it!\n");

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}